#include "hip/hip_runtime.h"
/*******************************************************
* Author R.Conway                                       *
*                                                       *
* Description: Use CUDA to convery a 2D array of RGB    *
*              pixels to a 2D array of grayscale pixels *
*                                                       *
********************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime_api.h>

//#include "GPUProcess.h"


/**
  * Function to compute the Keccak[r, c] sponge function over a given input.
  * @param  rate            The value of the rate r.
  * @param  capacity        The value of the capacity c.
  * @param  input           Pointer to the input message.
  * @param  inputByteLen    The number of input bytes provided in the input message.
  * @param  delimitedSuffix Bits that will be automatically appended to the end
  *                         of the input message, as in domain separation.
  *                         This is a byte containing from 0 to 7 bits
  *                         These <i>n</i> bits must be in the least significant bit positions
  *                         and must be delimited with a bit 1 at position <i>n</i>
  *                         (counting from 0=LSB to 7=MSB) and followed by bits 0
  *                         from position <i>n</i>+1 to position 7.
  *                         Some examples:
  *                             - If no bits are to be appended, then @a delimitedSuffix must be 0x01.
  *                             - If the 2-bit sequence 0,1 is to be appended (as for SHA3-*), @a delimitedSuffix must be 0x06.
  *                             - If the 4-bit sequence 1,1,1,1 is to be appended (as for SHAKE*), @a delimitedSuffix must be 0x1F.
  *                             - If the 7-bit sequence 1,1,0,1,0,0,0 is to be absorbed, @a delimitedSuffix must be 0x8B.
  * @param  output          Pointer to the buffer where to store the output.
  * @param  outputByteLen   The number of output bytes desired.
  * @pre    One must have r+c=1600 and the rate a multiple of 8 bits in this implementation.
  */
__global__ void Keccak_gpu(unsigned int rate, unsigned int capacity, uint64_t* input, int n_inputs);

typedef uint64_t tKeccakLane;

#define ROL64(a, offset) ((((uint64_t)a) << offset) ^ (((uint64_t)a) >> (64-offset)))
#define i(x, y) ((x)+5*(y))
#define Keccak_Rounds 24
#define uint_row_size 25*sizeof(uint64_t)
#define SHA3_CONST(x) x##L
enum last_permutation { Theta, Rho_PI, Chi, Iota };
char* perm_strings[4] = {(char *)"Theta", (char *)"Rho_PI", (char *)"Chi", (char *)"Iota" };
enum last_permutation last_perm;

// Works for Little Endian system, more needed to support Big Endian
#define readLane(x, y)          (((tKeccakLane*)state)[i(x, y)])
#define writeLane(x, y, lane)   (((tKeccakLane*)state)[i(x, y)]) = (lane)
#define XORLane(x, y, lane)     (((tKeccakLane*)state)[i(x, y)]) ^= (lane)



//void VerifyGPUOperation();

void GPUProcessing(uint64_t* input, int num_inputs)
{
    uint64_t* d_state_in;
    uint64_t* state_out;
    hipEvent_t start, stop;
    float time;
    int state_size_bytes = 8 * num_inputs * uint_row_size;

    dim3 dimGrid(ceil(num_inputs / 16.0), 1, 1);
    dim3 dimBlock(16, 16, 1);

    /* Allocate device memory */
    checkCudaErrors(hipMalloc((void**)&d_state_in, state_size_bytes));

    /* Allocate host memory for state_out */
    state_out = (uint64_t*)malloc(state_size_bytes);

    /* Initialise device memory */
    checkCudaErrors(hipMemcpy(d_state_in, state_out, state_size_bytes, hipMemcpyHostToDevice));

    /* Create timing events*/
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    /* Launch CUDA GPU kernels */
    Keccak_gpu <<<dimGrid, dimBlock >> > (256, 1344, d_state_in, num_inputs);
    printf("State out[0]: %.16llx", state_out[0]);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("\nTime for kernel execution is %3.2f ms\n", time);

    /* Copy from device back to host */
    checkCudaErrors(hipMemcpy(input, d_state_in, state_size_bytes, hipMemcpyDeviceToHost));

    /* Verify correct operation */
    // Ommitted here ...

    /* Free allocated memory on both device and host */
    checkCudaErrors(hipFree(d_state_in));
    printf(" Hashing complete \n");
}

__device__ static void print_state(uint64_t state[25], int round) {
    int x, y;
    printf("%s round %i\n", perm_strings[last_perm], round);
    //print in format of sha3 example pdf
    for (int p = 0; p < 5; p++) {
        for (int l = 0; l < 5; l++) {
            printf("[%i,%i]: %.16lx ", l, p, readLane(l, p));
            if (((p * 5 + l) % 2) == 1)
                printf("\n");
        }
    }
    printf("\n");
}


__device__ void KeccakF1600(void* state)
{
    unsigned int round, x, y, j, t;

    static const uint64_t keccakf_rndc[24] = {
    SHA3_CONST(0x0000000000000001UL), SHA3_CONST(0x0000000000008082UL),
    SHA3_CONST(0x800000000000808aUL), SHA3_CONST(0x8000000080008000UL),
    SHA3_CONST(0x000000000000808bUL), SHA3_CONST(0x0000000080000001UL),
    SHA3_CONST(0x8000000080008081UL), SHA3_CONST(0x8000000000008009UL),
    SHA3_CONST(0x000000000000008aUL), SHA3_CONST(0x0000000000000088UL),
    SHA3_CONST(0x0000000080008009UL), SHA3_CONST(0x000000008000000aUL),
    SHA3_CONST(0x000000008000808bUL), SHA3_CONST(0x800000000000008bUL),
    SHA3_CONST(0x8000000000008089UL), SHA3_CONST(0x8000000000008003UL),
    SHA3_CONST(0x8000000000008002UL), SHA3_CONST(0x8000000000000080UL),
    SHA3_CONST(0x000000000000800aUL), SHA3_CONST(0x800000008000000aUL),
    SHA3_CONST(0x8000000080008081UL), SHA3_CONST(0x8000000000008080UL),
    SHA3_CONST(0x0000000080000001UL), SHA3_CONST(0x8000000080008008UL)
    };

    // uint8_t LFSRstate = 0x01;
    //print_state(state,0);
    for (round = 0; round < 25; round++)
    {
        //print_state(state, round);
        //Theta step
        {
            tKeccakLane C[5], D;

            for (x = 0; x < 5; x++)
                C[x] = readLane(x, 0) ^ readLane(x, 1) ^ readLane(x, 2) ^ readLane(x, 3) ^ readLane(x, 4);
            for (x = 0; x < 5; x++) {
                D = C[(x + 4) % 5] ^ ROL64(C[(x + 1) % 5], 1);
                for (y = 0; y < 5; y++)
                    XORLane(x, y, D);
            }
            //last_perm = Theta;
            //print_state(state, round);
        }
        //Rho and Pi step
        {
            tKeccakLane current, tmp;

            //Start at (x,y) = (1,0)
            x = 1;
            y = 0;
            current = readLane(x, y);

            for (t = 0; t < 24; t++) {
                unsigned int r = ((t + 1) * (t + 2) / 2) % 64;
                unsigned int Y = (2 * x + 3 * y) % 5;
                x = y;
                y = Y;
                tmp = readLane(x, y);
                writeLane(x, y, ROL64(current, r));
                current = tmp;
            }
            //last_perm = Rho_PI;
            //print_state(state, round);
        }
        //Chi step
        {
            tKeccakLane temp[5];
            for (y = 0; y < 5; y++) {
                for (x = 0; x < 5; x++)
                    temp[x] = readLane(x, y);
                for (x = 0; x < 5; x++)
                    writeLane(x, y, temp[x] ^ ((~temp[(x + 1) % 5]) & temp[(x + 2) % 5]));
            }
            //last_perm = Chi;
            //print_state(state, round);
        }

        //Iota step
        {
            XORLane(0, 0, keccakf_rndc[round]);
            //last_perm = Iota;
            //print_state(state, round);
        }

    }

}

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAXSTRLEN sizeof("0xa3a3a3a3a3a3a3a3")

__global__ void Keccak_gpu(unsigned int rate, unsigned int capacity, uint64_t* input, int n_inputs)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    //Ensure rate and capacity sum to 1600 for keccakF1600
    if (((rate + capacity) != 1600) || ((rate % 8) != 0))
        return;
    
    if (c < n_inputs) {     
        KeccakF1600(&input[c]);
    }
}
